#include "type.cuh"
#include "SHA512.cuh"

//int main()
//{
//	hipEvent_t start, stop;
//	uint8_t pt[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
//	uint8_t salt[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
//	uint64_t DK[8];
//	uint8_t* cu_salt = NULL;
//	uint64_t* cu_DK = NULL;
//	float elapsed_time_ms = 0.0f;
//
//	hipMalloc((void**)&cu_salt, 8);
//	hipMemcpy(cu_salt, salt, 8 * sizeof(uint64_t), hipMemcpyHostToDevice);
//	hipMalloc((void**)&cu_DK, 8 * 8);
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord(start, 0);
//	for (int i = 0; i < 1; i++) {
//		KMU_PBKDF2v2 << <BLOCKFULL, THREADFULL >> > (129973, cu_salt, 8, cu_DK);
//	}
//	hipEventRecord(stop, 0);
//	hipDeviceSynchronize();
//	hipEventSynchronize(start);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&elapsed_time_ms, start, stop);
//	printf("Performance : %4.2f GB/s\n", elapsed_time_ms);
//}