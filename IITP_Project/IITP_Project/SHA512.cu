#include "hip/hip_runtime.h"
#include "type.cuh"
#include "SHA512.cuh"
//CPU part

__constant__ uint64_t cont_512[80] =
{
	0x428a2f98d728ae22, 0x7137449123ef65cd, 0xb5c0fbcfec4d3b2f, 0xe9b5dba58189dbbc,
	0x3956c25bf348b538, 0x59f111f1b605d019, 0x923f82a4af194f9b, 0xab1c5ed5da6d8118,
	0xd807aa98a3030242, 0x12835b0145706fbe, 0x243185be4ee4b28c, 0x550c7dc3d5ffb4e2,
	0x72be5d74f27b896f, 0x80deb1fe3b1696b1, 0x9bdc06a725c71235, 0xc19bf174cf692694,
	0xe49b69c19ef14ad2, 0xefbe4786384f25e3, 0x0fc19dc68b8cd5b5, 0x240ca1cc77ac9c65,
	0x2de92c6f592b0275, 0x4a7484aa6ea6e483, 0x5cb0a9dcbd41fbd4, 0x76f988da831153b5,
	0x983e5152ee66dfab, 0xa831c66d2db43210, 0xb00327c898fb213f, 0xbf597fc7beef0ee4,
	0xc6e00bf33da88fc2, 0xd5a79147930aa725, 0x06ca6351e003826f, 0x142929670a0e6e70,
	0x27b70a8546d22ffc, 0x2e1b21385c26c926, 0x4d2c6dfc5ac42aed, 0x53380d139d95b3df,
	0x650a73548baf63de, 0x766a0abb3c77b2a8, 0x81c2c92e47edaee6, 0x92722c851482353b,
	0xa2bfe8a14cf10364, 0xa81a664bbc423001, 0xc24b8b70d0f89791, 0xc76c51a30654be30,
	0xd192e819d6ef5218, 0xd69906245565a910, 0xf40e35855771202a, 0x106aa07032bbd1b8,
	0x19a4c116b8d2d0c8, 0x1e376c085141ab53, 0x2748774cdf8eeb99, 0x34b0bcb5e19b48a8,
	0x391c0cb3c5c95a63, 0x4ed8aa4ae3418acb, 0x5b9cca4f7763e373, 0x682e6ff3d6b2b8a3,
	0x748f82ee5defb2fc, 0x78a5636f43172f60, 0x84c87814a1f0ab72, 0x8cc702081a6439ec,
	0x90befffa23631e28, 0xa4506cebde82bde9, 0xbef9a3f7b2c67915,0xc67178f2e372532b,
	0xca273eceea26619c, 0xd186b8c721c0c207, 0xeada7dd6cde0eb1e, 0xf57d4f7fee6ed178,
	0x06f067aa72176fba, 0x0a637dc5a2c898a6, 0x113f9804bef90dae, 0x1b710b35131c471b,
	0x28db77f523047d84, 0x32caab7b40c72493, 0x3c9ebe0a15c9bebc, 0x431d67c49c100d4c,
	0x4cc5d4becb3e42b6, 0x597f299cfc657e2a, 0x5fcb6fab3ad6faec, 0x6c44198c4a475817
};

__device__ void KMU_PBKDF_SHA_BLOCK(const uint64_t* PT, uint64_t* output)
{
	volatile uint64_t w0_t = (PT[0]);
	volatile uint64_t w1_t = (PT[1]);
	volatile uint64_t w2_t = (PT[2]);
	volatile uint64_t w3_t = (PT[3]);
	volatile uint64_t w4_t = (PT[4]);
	volatile uint64_t w5_t = (PT[5]);
	volatile uint64_t w6_t = (PT[6]);
	volatile uint64_t w7_t = (PT[7]);
	volatile uint64_t w8_t = (PT[8]);
	volatile uint64_t w9_t = (PT[9]);
	volatile uint64_t wa_t = (PT[10]);
	volatile uint64_t wb_t = (PT[11]);
	volatile uint64_t wc_t = (PT[12]);
	volatile uint64_t wd_t = (PT[13]);
	volatile uint64_t we_t = (PT[14]);
	volatile uint64_t wf_t = (PT[15]);

	uint64_t a, b, c, d, e, f, g, h = 0;
	a = 0x6a09e667f3bcc908;
	b = 0xbb67ae8584caa73b;
	c = 0x3c6ef372fe94f82b;
	d = 0xa54ff53a5f1d36f1;
	e = 0x510e527fade682d1;
	f = 0x9b05688c2b3e6c1f;
	g = 0x1f83d9abfb41bd6b;
	h = 0x5be0cd19137e2179;

	#define ROUND_EXPAND()								\
	{													\
		w0_t = SHA512_EXPAND (we_t, w9_t, w1_t, w0_t);  \
		w1_t = SHA512_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
		w2_t = SHA512_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
		w3_t = SHA512_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
		w4_t = SHA512_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
		w5_t = SHA512_EXPAND (w3_t, we_t, w6_t, w5_t);  \
		w6_t = SHA512_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
		w7_t = SHA512_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
		w8_t = SHA512_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
		w9_t = SHA512_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
		wa_t = SHA512_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
		wb_t = SHA512_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
		wc_t = SHA512_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
		wd_t = SHA512_EXPAND (wb_t, w6_t, we_t, wd_t);  \
		we_t = SHA512_EXPAND (wc_t, w7_t, wf_t, we_t);  \
		wf_t = SHA512_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
	}

	#define ROUND_STEP(i)																	\
	{																						\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w0_t, cont_512[i +  0]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w1_t, cont_512[i +  1]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, w2_t, cont_512[i +  2]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, w3_t, cont_512[i +  3]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, w4_t, cont_512[i +  4]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, w5_t, cont_512[i +  5]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, w6_t, cont_512[i +  6]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, w7_t, cont_512[i +  7]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w8_t, cont_512[i +  8]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w9_t, cont_512[i +  9]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, wa_t, cont_512[i + 10]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, wb_t, cont_512[i + 11]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, wc_t, cont_512[i + 12]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, wd_t, cont_512[i + 13]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, we_t, cont_512[i + 14]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, wf_t, cont_512[i + 15]);	\
	}
	ROUND_STEP(0);
	for (int i = 16; i < 80; i += 16) {
		ROUND_EXPAND(); 
		ROUND_STEP(i);
	}
	output[0] = a + 0x6a09e667f3bcc908;
	output[1] = b + 0xbb67ae8584caa73b;
	output[2] = c + 0x3c6ef372fe94f82b;
	output[3] = d + 0xa54ff53a5f1d36f1;
	output[4] = e + 0x510e527fade682d1;
	output[5] = f + 0x9b05688c2b3e6c1f;
	output[6] = g + 0x1f83d9abfb41bd6b;
	output[7] = h + 0x5be0cd19137e2179;
}

__device__ void KMU_PBKDF2_BLOCK_CORE(uint64_t* PRE_IN, uint64_t* hash, uint64_t ptLen, uint64_t* OUT)
{
	uint64_t a, b, c, d, e, f, g, h = 0;
	a = PRE_IN[0];
	b = PRE_IN[1];
	c = PRE_IN[2];
	d = PRE_IN[3];
	e = PRE_IN[4];
	f = PRE_IN[5];
	g = PRE_IN[6];
	h = PRE_IN[7];

	//Padding
	volatile uint64_t w0_t = hash[0];
	volatile uint64_t w1_t = hash[1];
	volatile uint64_t w2_t = hash[2];
	volatile uint64_t w3_t = hash[3];
	volatile uint64_t w4_t = hash[4];
	volatile uint64_t w5_t = hash[5];
	volatile uint64_t w6_t = hash[6];
	volatile uint64_t w7_t = hash[7];
	volatile uint64_t w8_t = 0x8000000000000000;
	volatile uint64_t w9_t = 0;
	volatile uint64_t wa_t = 0;
	volatile uint64_t wb_t = 0;
	volatile uint64_t wc_t = 0;
	volatile uint64_t wd_t = 0;
	volatile uint64_t we_t = 0;
	volatile uint64_t wf_t = ptLen  << 3;
#define ROUND_EXPAND()									\
	{													\
		w0_t = SHA512_EXPAND (we_t, w9_t, w1_t, w0_t);  \
		w1_t = SHA512_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
		w2_t = SHA512_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
		w3_t = SHA512_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
		w4_t = SHA512_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
		w5_t = SHA512_EXPAND (w3_t, we_t, w6_t, w5_t);  \
		w6_t = SHA512_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
		w7_t = SHA512_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
		w8_t = SHA512_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
		w9_t = SHA512_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
		wa_t = SHA512_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
		wb_t = SHA512_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
		wc_t = SHA512_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
		wd_t = SHA512_EXPAND (wb_t, w6_t, we_t, wd_t);  \
		we_t = SHA512_EXPAND (wc_t, w7_t, wf_t, we_t);  \
		wf_t = SHA512_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
	}

#define ROUND_STEP(i)																		\
	{																						\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w0_t, cont_512[i +  0]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w1_t, cont_512[i +  1]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, w2_t, cont_512[i +  2]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, w3_t, cont_512[i +  3]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, w4_t, cont_512[i +  4]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, w5_t, cont_512[i +  5]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, w6_t, cont_512[i +  6]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, w7_t, cont_512[i +  7]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w8_t, cont_512[i +  8]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w9_t, cont_512[i +  9]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, wa_t, cont_512[i + 10]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, wb_t, cont_512[i + 11]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, wc_t, cont_512[i + 12]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, wd_t, cont_512[i + 13]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, we_t, cont_512[i + 14]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, wf_t, cont_512[i + 15]);	\
	}
	ROUND_STEP(0);
	for (int i = 16; i < 80; i += 16) {
		ROUND_EXPAND();
		ROUND_STEP(i);
	}

	OUT[0] = a + PRE_IN[0];
	OUT[1] = b + PRE_IN[1];
	OUT[2] = c + PRE_IN[2];
	OUT[3] = d + PRE_IN[3];
	OUT[4] = e + PRE_IN[4];
	OUT[5] = f + PRE_IN[5];
	OUT[6] = g + PRE_IN[6];
	OUT[7] = h + PRE_IN[7];
}

__device__ void KMU_PRE_HMAC_SHA512(uint8_t* pt, uint64_t ptLen, uint64_t* OPAD_out, uint64_t* IPAD_out)
{
	uint8_t IPAD[128] = { 0x36, };
	uint8_t OPAD[128] = { 0x5c, };
	for (int i = 0; i < 128; i++) {
		IPAD[i] = 0x36;
		OPAD[i] = 0x5c;
	}
	uint64_t buffer[16];
	for (int i = 0; i < ptLen; i++) {
		IPAD[i] = IPAD[i] ^ pt[i];
		OPAD[i] = OPAD[i] ^ pt[i];
	}
	for (int i = 0; i < 16; i++) {
		*(uint64_t*)(IPAD + (i << 3)) = ENDIAN_CHANGE(*(uint64_t*)(IPAD + (i << 3)));
		*(uint64_t*)(OPAD + (i << 3)) = ENDIAN_CHANGE(*(uint64_t*)(IPAD + (i << 3)));
	}
	KMU_PBKDF_SHA_BLOCK((uint64_t*)IPAD, IPAD_out);
	KMU_PBKDF_SHA_BLOCK((uint64_t*)OPAD, OPAD_out);


}

__device__ void KMU_PBKDF2_CORE(uint8_t* pt, uint64_t ptLen, uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* DK) {
	uint64_t dk[8] = { 0, };
	uint64_t IPAD_out[8] = { 0, };
	uint64_t OPAD_out[8] = { 0, };
	uint64_t out[8] = { 0, };
	uint64_t Buffer[8] = { 0, };

	KMU_PRE_HMAC_SHA512(pt, ptLen, IPAD_out, OPAD_out);

	//U0 operation
	uint8_t buffer[64] = { 0, };
	for (int i = 0; i < saLen; i++) {
		buffer[i] = salt[i];
	}
	
	buffer[saLen + 0] = (1 << 24) & 0xff;
	buffer[saLen + 1] = (1 << 16) & 0xff;
	buffer[saLen + 2] = (1 << 8) & 0xff;
	buffer[saLen + 3] = (1) & 0xff;

	for (int i = 0; i < 8; i++) {
		*(uint64_t*)(buffer + (i << 3)) = ENDIAN_CHANGE(*(uint64_t*)(buffer + (i << 3)));
	}

	KMU_PBKDF2_BLOCK_CORE(IPAD_out, (uint64_t*)buffer, 64 + saLen + 4, dk);
	KMU_PBKDF2_BLOCK_CORE(OPAD_out, dk, 64, Buffer);

	for (int i = 1; i < iteration; i++) {
		KMU_PBKDF2_BLOCK_CORE(IPAD_out, Buffer, 128, dk);
		KMU_PBKDF2_BLOCK_CORE(OPAD_out, dk, 128, Buffer);
		for (int i = 0; i < 8; i++)
			out[i] ^= Buffer[i];
	}
	
	for (int i = 0; i < 8; i++) {
		DK[i] = out[i];
	}
}

__global__ void KMU_PBKDF2(uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* DK)
{
	uint8_t pt[4] = { 0, };
	pt[0] = blockIdx.x;
	pt[1] = threadIdx.x;
	uint64_t out[8];
	for (int i = 0; i < 256 * 256; i++) {
		pt[2] = i / 256;
		pt[3] = i % 256;
	}
	KMU_PBKDF2_CORE(pt, 4, 129973, salt, saLen, out);

	for (int i = 0; i < 8; i++) {
		DK[i] = out[i];
	}
	if (blockIdx.x == 0 && threadIdx.x == 0) {
		for (int i = 0; i < 8; i++) {
			printf("%016llx ", DK[i]);
		}
		printf("\n");
	}
}

int main()
{
	hipEvent_t start, stop;
	uint8_t pt[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
	uint8_t salt[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
	uint64_t DK[8];
	uint8_t* cu_salt = NULL;
	uint64_t* cu_DK = NULL;
	float elapsed_time_ms = 0.0f;

	hipMalloc((void**)&cu_salt, 8);
	hipMemcpy(cu_salt, salt, 8, hipMemcpyHostToDevice);
	hipMalloc((void**)&cu_DK, 8 * 8);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (int i = 0; i < 1; i++) {
		KMU_PBKDF2 << <256, 256 >> > (129973, cu_salt, 8, cu_DK);
	}
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("Performance : %4.2f GB/s\n", elapsed_time_ms);
}

//int main()
//{
//	uint64_t A[16] = {0, };
//	A[0] = 0x6162636465666768;
//	A[1] = 0x6263646566676869;
//	A[2] = 0x636465666768696a;
//	A[3] = 0x6465666768696a6b;
//	A[4] = 0x65666768696a6b6c;
//	A[5] = 0x666768696a6b6c6d;
//	A[6] = 0x6768696a6b6c6d6e;
//	A[7] = 0x68696a6b6c6d6e6f;
//	A[8] = 0x696a6b6c6d6e6f70;
//	A[9] = 0x6a6b6c6d6e6f7071;
//	A[10] = 0x6b6c6d6e6f707172;
//	A[11] = 0x6c6d6e6f70717273;
//	A[12] = 0x6d6e6f7071727374;
//	A[13] = 0x6e6f707172737475;
//	A[14] = 0x8000000000000000;
//	A[15] = 0x0000000000000000;
//
//	uint64_t B[8] = { 0, };
//	uint64_t* cu_A = NULL;
//	uint64_t* cu_out = NULL;
//	uint64_t* cu_B = NULL;
//	uint64_t* cu_pre_IN = NULL;
//	uint64_t Pre[8];
//	Pre[0] = 0x4319017a2b706e69;
//	Pre[1] = 0xcd4b05938bae5e89;
//	Pre[2] = 0x0186bf199f30aa95;
//	Pre[3] = 0x6ef8b71d2f810585;
//	Pre[4] = 0xd787d6764b20bda2;
//	Pre[5] = 0xa260144709736920;
//	Pre[6] = 0x00ec057f37d14b8e;
//	Pre[7] = 0x06add5b50e671c72;
//	hipMalloc((void**)&cu_A, 128);
//	hipMalloc((void**)&cu_B, 64);
//	hipMalloc((void**)&cu_out, 64);
//	hipMalloc((void**)&cu_pre_IN, 64);
//	hipMemcpy(cu_A, A, 128, hipMemcpyHostToDevice);
//	hipMemcpy(cu_B, B, 64, hipMemcpyHostToDevice);
//	hipMemcpy(cu_pre_IN, Pre, 64, hipMemcpyHostToDevice);
//	//KMU_PBKDF_SHA_BLOCK << <1, 1 >> > (cu_A, cu_out);
//	//KMU_PBKDF2_BLOCK_CORE << <1, 1 >> > (cu_pre_IN, cu_B, 112, cu_out);
//	KMU_PRE_HMAC_SHA512 << <1, 1 >> > (cu_out);
//	hipMemcpy(B, cu_out, 64, hipMemcpyDeviceToHost);
//	for (int i = 0; i < 8; i++)
//		printf("%016llx ", B[i]);
//
//}

//
//__device__ void KMU_PBKDF2_BLOCK_CORE(uint64_t* PRE_IN, uint64_t* hash, uint64_t ptLen, uint64_t* OUT)
//{
//	uint64_t a, b, c, d, e, f, g, h = 0;
//	a = PRE_IN[0];
//	b = PRE_IN[1];
//	c = PRE_IN[2];
//	d = PRE_IN[3];
//	e = PRE_IN[4];
//	f = PRE_IN[5];
//	g = PRE_IN[6];
//	h = PRE_IN[7];
//
//	//Padding
//	uint64_t w0_t = hash[0];
//	uint64_t w1_t = hash[1];
//	uint64_t w2_t = hash[2];
//	uint64_t w3_t = hash[3];
//	uint64_t w4_t = hash[4];
//	uint64_t w5_t = hash[5];
//	uint64_t w6_t = hash[6];
//	uint64_t w7_t = hash[7];
//	uint64_t w8_t = 0x8000000000000000;
//	uint64_t w9_t = 0;
//	uint64_t wa_t = 0;
//	uint64_t wb_t = 0;
//	uint64_t wc_t = 0;
//	uint64_t wd_t = 0;
//	uint64_t we_t = 0;
//	uint64_t wf_t = ptLen  << 3;
//
//#define ROUND_EXPAND()									\
//	{													\
//		w0_t = SHA512_EXPAND (we_t, w9_t, w1_t, w0_t);  \
//		w1_t = SHA512_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
//		w2_t = SHA512_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
//		w3_t = SHA512_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
//		w4_t = SHA512_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
//		w5_t = SHA512_EXPAND (w3_t, we_t, w6_t, w5_t);  \
//		w6_t = SHA512_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
//		w7_t = SHA512_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
//		w8_t = SHA512_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
//		w9_t = SHA512_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
//		wa_t = SHA512_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
//		wb_t = SHA512_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
//		wc_t = SHA512_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
//		wd_t = SHA512_EXPAND (wb_t, w6_t, we_t, wd_t);  \
//		we_t = SHA512_EXPAND (wc_t, w7_t, wf_t, we_t);  \
//		wf_t = SHA512_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
//	}
//
//#define ROUND_STEP(i)																		\
//	{																						\
//		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w0_t, cont_512[i +  0]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w1_t, cont_512[i +  1]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, w2_t, cont_512[i +  2]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, w3_t, cont_512[i +  3]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, w4_t, cont_512[i +  4]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, w5_t, cont_512[i +  5]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, w6_t, cont_512[i +  6]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, w7_t, cont_512[i +  7]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w8_t, cont_512[i +  8]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w9_t, cont_512[i +  9]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, wa_t, cont_512[i + 10]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, wb_t, cont_512[i + 11]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, wc_t, cont_512[i + 12]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, wd_t, cont_512[i + 13]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, we_t, cont_512[i + 14]);	\
//		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, wf_t, cont_512[i + 15]);	\
//	}
//	ROUND_STEP(0);
//	for (int i = 16; i < 80; i += 16) {
//		ROUND_EXPAND();
//		ROUND_STEP(i);
//	}
//
//	OUT[0] = a + 0x6a09e667f3bcc908;
//	OUT[1] = b + 0xbb67ae8584caa73b;
//	OUT[2] = c + 0x3c6ef372fe94f82b;
//	OUT[3] = d + 0xa54ff53a5f1d36f1;
//	OUT[4] = e + 0x510e527fade682d1;
//	OUT[5] = f + 0x9b05688c2b3e6c1f;
//	OUT[6] = g + 0x1f83d9abfb41bd6b;
//	OUT[7] = h + 0x5be0cd19137e2179;
//}
//
//__device__ void KMU_PRE_HMAC_SHA512(uint8_t* pt, uint64_t ptLen, uint64_t* IPAD_out, uint64_t* OPAD_out)
//{
//	uint8_t IPAD[128] = { 0x36, };
//	uint8_t OPAD[128] = { 0x5c, };
//	uint64_t buffer[16];
//	for (int i = 0; i < ptLen; i++) {
//		IPAD[i] = IPAD[i] ^ pt[i];
//		OPAD[i] = OPAD[i] ^ pt[i];
//	}
//	KMU_PBKDF_SHA_BLOCK((uint64_t*)IPAD, IPAD_out);
//	KMU_PBKDF_SHA_BLOCK((uint64_t*)OPAD, OPAD_out);
//
//}
//
////Fixed dkLen = 64byte
//__device__ void KMU_PBKDF2_CORE(uint8_t* pt, uint64_t ptLen, uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* DK) {
//	uint64_t dk[8] = { 0, };
//	uint64_t IPAD_out[8] = { 0, };
//	uint64_t OPAD_out[8] = { 0, };
//	uint64_t out[8] = { 0, };
//	KMU_PRE_HMAC_SHA512(pt, ptLen, IPAD_out, OPAD_out);
//
//	//U0 operation
//	uint8_t buffer[64] = { 0, };
//	for (int i = 0; i < saLen; i++) {
//		buffer[i] = salt[i];
//	}
//	buffer[saLen + 0] = (1 >> 24) & 0xff;
//	buffer[saLen + 1] = (1 >> 16) & 0xff;
//	buffer[saLen + 2] = (1 >> 8) & 0xff;
//	buffer[saLen + 3] = (1) & 0xff;
//	KMU_PBKDF2_BLOCK_CORE(IPAD_out, (uint64_t*)buffer, 64 + saLen + 4, dk);
//	KMU_PBKDF2_BLOCK_CORE(OPAD_out, dk, 64, dk);
//	for (int i = 0; i < 8; i++)
//		out[i] = dk[i];
//	for (int i = 1; i < iteration; i++) {
//		KMU_PBKDF2_BLOCK_CORE(IPAD_out, dk, 128, dk);
//		KMU_PBKDF2_BLOCK_CORE(OPAD_out, dk, 128, dk);
//		for (int i = 0; i < 8; i++)
//			out[i] ^= dk[i];
//	}
//	
//	for (int i = 0; i < 8; i++) {
//		DK[i] = out[i];
//	}
//}
//
//
////passwordLen = 4
//__global__ void KMU_PBKDF2(uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* DK)
//{
//	uint8_t pt[4] = { 0, };
//	pt[0] = blockIdx.x;
//	pt[1] = threadIdx.x;
//	uint64_t out[8];
//	for (int i = 0; i < 256 * 256; i++) {
//		pt[2] = i / 256;
//		pt[3] = i % 256;
//	}
//	KMU_PBKDF2_CORE(pt, 4, 129973, salt, saLen, out);
//
//	for (int i = 0; i < 8; i++) {
//		DK[i] = out[i];
//	}
//	if (blockIdx.x == 0 && threadIdx.x == 0) {
//		for (int i = 0; i < 8; i++) {
//			printf("%016llu ", DK[i]);
//		}
//		printf("\n");
//	}
//
//}
//
//int main()
//{
//	hipEvent_t start, stop;
//	uint8_t pt[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
//	uint8_t salt[8] = { 0, 1, 2, 3, 4, 5, 6, 7 };
//	uint64_t DK[8];
//	uint8_t* cu_salt = NULL;
//	uint64_t* cu_DK = NULL;
//	float elapsed_time_ms = 0.0f;
//
//	hipMalloc((void**)&cu_salt, 8);
//	hipMemcpy(cu_salt, salt, 8, hipMemcpyHostToDevice);
//	hipMalloc((void**)&cu_DK, 8 * 8);
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord(start, 0);
//	for (int i = 0; i < 1; i++) {
//		KMU_PBKDF2 << <256, 256 >> > (129973, cu_salt, 8, cu_DK);
//	}
//	hipEventRecord(stop, 0);
//	hipDeviceSynchronize();
//	hipEventSynchronize(start);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&elapsed_time_ms, start, stop);
//	printf("Performance : %4.2f GB/s\n", elapsed_time_ms);
//}