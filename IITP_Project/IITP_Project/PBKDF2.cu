#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STEAM
#include "type.cuh"
#include "SHA512.cuh"

//Clear
__constant__ uint64_t cont_512v2[80] =
{
	0x428a2f98d728ae22, 0x7137449123ef65cd, 0xb5c0fbcfec4d3b2f, 0xe9b5dba58189dbbc,
	0x3956c25bf348b538, 0x59f111f1b605d019, 0x923f82a4af194f9b, 0xab1c5ed5da6d8118,
	0xd807aa98a3030242, 0x12835b0145706fbe, 0x243185be4ee4b28c, 0x550c7dc3d5ffb4e2,
	0x72be5d74f27b896f, 0x80deb1fe3b1696b1, 0x9bdc06a725c71235, 0xc19bf174cf692694,
	0xe49b69c19ef14ad2, 0xefbe4786384f25e3, 0x0fc19dc68b8cd5b5, 0x240ca1cc77ac9c65,
	0x2de92c6f592b0275, 0x4a7484aa6ea6e483, 0x5cb0a9dcbd41fbd4, 0x76f988da831153b5,
	0x983e5152ee66dfab, 0xa831c66d2db43210, 0xb00327c898fb213f, 0xbf597fc7beef0ee4,
	0xc6e00bf33da88fc2, 0xd5a79147930aa725, 0x06ca6351e003826f, 0x142929670a0e6e70,
	0x27b70a8546d22ffc, 0x2e1b21385c26c926, 0x4d2c6dfc5ac42aed, 0x53380d139d95b3df,
	0x650a73548baf63de, 0x766a0abb3c77b2a8, 0x81c2c92e47edaee6, 0x92722c851482353b,
	0xa2bfe8a14cf10364, 0xa81a664bbc423001, 0xc24b8b70d0f89791, 0xc76c51a30654be30,
	0xd192e819d6ef5218, 0xd69906245565a910, 0xf40e35855771202a, 0x106aa07032bbd1b8,
	0x19a4c116b8d2d0c8, 0x1e376c085141ab53, 0x2748774cdf8eeb99, 0x34b0bcb5e19b48a8,
	0x391c0cb3c5c95a63, 0x4ed8aa4ae3418acb, 0x5b9cca4f7763e373, 0x682e6ff3d6b2b8a3,
	0x748f82ee5defb2fc, 0x78a5636f43172f60, 0x84c87814a1f0ab72, 0x8cc702081a6439ec,
	0x90befffa23631e28, 0xa4506cebde82bde9, 0xbef9a3f7b2c67915, 0xc67178f2e372532b,
	0xca273eceea26619c, 0xd186b8c721c0c207, 0xeada7dd6cde0eb1e, 0xf57d4f7fee6ed178,
	0x06f067aa72176fba, 0x0a637dc5a2c898a6, 0x113f9804bef90dae, 0x1b710b35131c471b,
	0x28db77f523047d84, 0x32caab7b40c72493, 0x3c9ebe0a15c9bebc, 0x431d67c49c100d4c,
	0x4cc5d4becb3e42b6, 0x597f299cfc657e2a, 0x5fcb6fab3ad6faec, 0x6c44198c4a475817
};

__constant__ uint64_t TT[1] = { 0x2527B51365102B38 };

//Clear
__device__ void KMU_PBKDF_SHA_BLOCKv2(uint64_t* PT, uint64_t* output)
{
	volatile uint64_t w0_t = (PT[0]);
	volatile uint64_t w1_t = (PT[1]);
	volatile uint64_t w2_t = (PT[2]);
	volatile uint64_t w3_t = (PT[3]);
	volatile uint64_t w4_t = (PT[4]);
	volatile uint64_t w5_t = (PT[5]);
	volatile uint64_t w6_t = (PT[6]);
	volatile uint64_t w7_t = (PT[7]);
	volatile uint64_t w8_t = (PT[8]);
	volatile uint64_t w9_t = (PT[9]);
	volatile uint64_t wa_t = (PT[10]);
	volatile uint64_t wb_t = (PT[11]);
	volatile uint64_t wc_t = (PT[12]);
	volatile uint64_t wd_t = (PT[13]);
	volatile uint64_t we_t = (PT[14]);
	volatile uint64_t wf_t = (PT[15]);

	uint64_t a, b, c, d, e, f, g, h = 0;
	a = 0x6a09e667f3bcc908;
	b = 0xbb67ae8584caa73b;
	c = 0x3c6ef372fe94f82b;
	d = 0xa54ff53a5f1d36f1;
	e = 0x510e527fade682d1;
	f = 0x9b05688c2b3e6c1f;
	g = 0x1f83d9abfb41bd6b;
	h = 0x5be0cd19137e2179;

#define ROUND_EXPAND()									\
	{													\
		w0_t = SHA512_EXPAND (we_t, w9_t, w1_t, w0_t);  \
		w1_t = SHA512_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
		w2_t = SHA512_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
		w3_t = SHA512_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
		w4_t = SHA512_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
		w5_t = SHA512_EXPAND (w3_t, we_t, w6_t, w5_t);  \
		w6_t = SHA512_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
		w7_t = SHA512_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
		w8_t = SHA512_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
		w9_t = SHA512_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
		wa_t = SHA512_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
		wb_t = SHA512_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
		wc_t = SHA512_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
		wd_t = SHA512_EXPAND (wb_t, w6_t, we_t, wd_t);  \
		we_t = SHA512_EXPAND (wc_t, w7_t, wf_t, we_t);  \
		wf_t = SHA512_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
	}

#define ROUND_STEP(i)																			\
	{																							\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w0_t, cont_512v2[i +  0]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w1_t, cont_512v2[i +  1]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, w2_t, cont_512v2[i +  2]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, w3_t, cont_512v2[i +  3]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, w4_t, cont_512v2[i +  4]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, w5_t, cont_512v2[i +  5]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, w6_t, cont_512v2[i +  6]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, w7_t, cont_512v2[i +  7]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w8_t, cont_512v2[i +  8]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w9_t, cont_512v2[i +  9]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, wa_t, cont_512v2[i + 10]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, wb_t, cont_512v2[i + 11]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, wc_t, cont_512v2[i + 12]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, wd_t, cont_512v2[i + 13]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, we_t, cont_512v2[i + 14]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, wf_t, cont_512v2[i + 15]);	\
	}
	ROUND_STEP(0);
	for (int i = 16; i < 80; i += 16) {
		ROUND_EXPAND();
		ROUND_STEP(i);
	}
	output[0] = a + 0x6a09e667f3bcc908;
	output[1] = b + 0xbb67ae8584caa73b;
	output[2] = c + 0x3c6ef372fe94f82b;
	output[3] = d + 0xa54ff53a5f1d36f1;
	output[4] = e + 0x510e527fade682d1;
	output[5] = f + 0x9b05688c2b3e6c1f;
	output[6] = g + 0x1f83d9abfb41bd6b;
	output[7] = h + 0x5be0cd19137e2179;
}

__device__ void KMU_PBKDF_SHA_BLOCK_COREv2(uint64_t* PREIN, uint64_t* hash, uint64_t ptLen, uint64_t* OUT) {
	
	uint64_t a = PREIN[0];
	uint64_t b = PREIN[1];
	uint64_t c = PREIN[2];
	uint64_t d = PREIN[3];
	uint64_t e = PREIN[4];
	uint64_t f = PREIN[5];
	uint64_t g = PREIN[6];
	uint64_t h = PREIN[7];

	//word setting
	volatile uint64_t w0_t = (hash[0]);
	volatile uint64_t w1_t = (hash[1]);
	volatile uint64_t w2_t = (hash[2]);
	volatile uint64_t w3_t = (hash[3]);
	volatile uint64_t w4_t = (hash[4]);
	volatile uint64_t w5_t = (hash[5]);
	volatile uint64_t w6_t = (hash[6]);
	volatile uint64_t w7_t = (hash[7]);
	volatile uint64_t w8_t = (0x8000000000000000);
	volatile uint64_t w9_t = 0;
	volatile uint64_t wa_t = 0;
	volatile uint64_t wb_t = 0;
	volatile uint64_t wc_t = 0;
	volatile uint64_t wd_t = 0;
	volatile uint64_t we_t = 0;
	volatile uint64_t wf_t = (ptLen << 3);


#define ROUND_EXPAND()									\
	{													\
		w0_t = SHA512_EXPAND (we_t, w9_t, w1_t, w0_t);  \
		w1_t = SHA512_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
		w2_t = SHA512_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
		w3_t = SHA512_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
		w4_t = SHA512_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
		w5_t = SHA512_EXPAND (w3_t, we_t, w6_t, w5_t);  \
		w6_t = SHA512_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
		w7_t = SHA512_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
		w8_t = SHA512_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
		w9_t = SHA512_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
		wa_t = SHA512_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
		wb_t = SHA512_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
		wc_t = SHA512_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
		wd_t = SHA512_EXPAND (wb_t, w6_t, we_t, wd_t);  \
		we_t = SHA512_EXPAND (wc_t, w7_t, wf_t, we_t);  \
		wf_t = SHA512_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
	}

#define ROUND_STEP(i)																			\
	{																							\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w0_t, cont_512v2[i +  0]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w1_t, cont_512v2[i +  1]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, w2_t, cont_512v2[i +  2]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, w3_t, cont_512v2[i +  3]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, w4_t, cont_512v2[i +  4]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, w5_t, cont_512v2[i +  5]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, w6_t, cont_512v2[i +  6]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, w7_t, cont_512v2[i +  7]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w8_t, cont_512v2[i +  8]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w9_t, cont_512v2[i +  9]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, wa_t, cont_512v2[i + 10]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, wb_t, cont_512v2[i + 11]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, wc_t, cont_512v2[i + 12]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, wd_t, cont_512v2[i + 13]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, we_t, cont_512v2[i + 14]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, wf_t, cont_512v2[i + 15]);	\
	}
	ROUND_STEP(0);
	for (int i = 16; i < 80; i += 16) {
		ROUND_EXPAND();
		ROUND_STEP(i);
	}

	OUT[0] = a + PREIN[0];
	OUT[1] = b + PREIN[1];
	OUT[2] = c + PREIN[2];
	OUT[3] = d + PREIN[3];
	OUT[4] = e + PREIN[4];
	OUT[5] = f + PREIN[5];
	OUT[6] = g + PREIN[6];
	OUT[7] = h + PREIN[7];
}

__device__ void KMU_PBKDF_SALTv1(uint64_t* PREIN, uint64_t* salt, uint64_t* OUT) {

	uint64_t a = PREIN[0];
	uint64_t b = PREIN[1];
	uint64_t c = PREIN[2];
	uint64_t d = PREIN[3];
	uint64_t e = PREIN[4];
	uint64_t f = PREIN[5];
	uint64_t g = PREIN[6];
	uint64_t h = PREIN[7];

	//word setting
	volatile uint64_t w0_t = ENDIAN_CHANGE(salt[0]);
	volatile uint64_t w1_t = ENDIAN_CHANGE(salt[1]);
	volatile uint64_t w2_t = ENDIAN_CHANGE(salt[2]);
	volatile uint64_t w3_t = ENDIAN_CHANGE(salt[3]);
	volatile uint64_t w4_t = ENDIAN_CHANGE(salt[4]);
	volatile uint64_t w5_t = ENDIAN_CHANGE(salt[5]);
	volatile uint64_t w6_t = ENDIAN_CHANGE(salt[6]);
	volatile uint64_t w7_t = ENDIAN_CHANGE(salt[7]);
	volatile uint64_t w8_t = ENDIAN_CHANGE(salt[8]);
	volatile uint64_t w9_t = ENDIAN_CHANGE(salt[9]);
	volatile uint64_t wa_t = ENDIAN_CHANGE(salt[10]);
	volatile uint64_t wb_t = ENDIAN_CHANGE(salt[11]);
	volatile uint64_t wc_t = ENDIAN_CHANGE(salt[12]);
	volatile uint64_t wd_t = ENDIAN_CHANGE(salt[13]);
	volatile uint64_t we_t = ENDIAN_CHANGE(salt[14]);
	volatile uint64_t wf_t = (salt[15]);

#define ROUND_EXPAND()									\
	{													\
		w0_t = SHA512_EXPAND (we_t, w9_t, w1_t, w0_t);  \
		w1_t = SHA512_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
		w2_t = SHA512_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
		w3_t = SHA512_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
		w4_t = SHA512_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
		w5_t = SHA512_EXPAND (w3_t, we_t, w6_t, w5_t);  \
		w6_t = SHA512_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
		w7_t = SHA512_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
		w8_t = SHA512_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
		w9_t = SHA512_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
		wa_t = SHA512_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
		wb_t = SHA512_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
		wc_t = SHA512_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
		wd_t = SHA512_EXPAND (wb_t, w6_t, we_t, wd_t);  \
		we_t = SHA512_EXPAND (wc_t, w7_t, wf_t, we_t);  \
		wf_t = SHA512_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
	}

#define ROUND_STEP(i)																			\
	{																							\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w0_t, cont_512v2[i +  0]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w1_t, cont_512v2[i +  1]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, w2_t, cont_512v2[i +  2]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, w3_t, cont_512v2[i +  3]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, w4_t, cont_512v2[i +  4]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, w5_t, cont_512v2[i +  5]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, w6_t, cont_512v2[i +  6]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, w7_t, cont_512v2[i +  7]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, a, b, c, d, e, f, g, h, w8_t, cont_512v2[i +  8]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, h, a, b, c, d, e, f, g, w9_t, cont_512v2[i +  9]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, g, h, a, b, c, d, e, f, wa_t, cont_512v2[i + 10]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, f, g, h, a ,b, c, d, e, wb_t, cont_512v2[i + 11]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, e, f, g, h, a ,b, c, d, wc_t, cont_512v2[i + 12]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, d, e, f, g, h, a ,b, c, wd_t, cont_512v2[i + 13]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, c, d, e, f, g, h, a, b, we_t, cont_512v2[i + 14]);	\
		SHA512_STEP(SHA512_F0, SHA512_F1, b, c, d, e, f, g, h, a, wf_t, cont_512v2[i + 15]);	\
	}
	ROUND_STEP(0);
	for (int i = 16; i < 80; i += 16) {
		ROUND_EXPAND();
		ROUND_STEP(i);
	}

	OUT[0] = a + PREIN[0];
	OUT[1] = b + PREIN[1];
	OUT[2] = c + PREIN[2];
	OUT[3] = d + PREIN[3];
	OUT[4] = e + PREIN[4];
	OUT[5] = f + PREIN[5];
	OUT[6] = g + PREIN[6];
	OUT[7] = h + PREIN[7];
}

__device__ void KMU_PRE_HMAC_SHA512v2(uint64_t pt, uint64_t ptLen, uint64_t* OPAD_out, uint64_t* IPAD_out)
{
	
	uint64_t IPAD[16];
	uint64_t OPAD[16];
	for (int i = 0; i < 16; i++) {
		IPAD[i] = 0x3636363636363636;
		OPAD[i] = 0x5c5c5c5c5c5c5c5c;
	}
	
	//���� password word�� ������ ���� ��� �����ϱ�
	for (int i = 0; i < 1; i++) {
		IPAD[i] = IPAD[i] ^ pt;
		OPAD[i] = OPAD[i] ^ pt;
	}
	
	KMU_PBKDF_SHA_BLOCKv2(IPAD, IPAD_out);
	KMU_PBKDF_SHA_BLOCKv2(OPAD, OPAD_out);
}

__device__ void KMU_PKBDF2_Core_v2(uint64_t pt, uint64_t ptLen, uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* DK, uint8_t* cracking) {
	uint64_t OPAD_out[16];
	uint64_t IPAD_out[16];
	uint64_t dk1[8];
	uint64_t dk2[8];
	uint64_t out[8];
	uint8_t buffer[128] = { 0, };

	//pre_computation
	uint64_t crackpt = pt;
	KMU_PRE_HMAC_SHA512v2(crackpt, ptLen, OPAD_out, IPAD_out);

	//salt value gen
	for (int i = 0; i < saLen; i++)
		buffer[i] = salt[i];
	buffer[saLen + 3] = (1) & 0xff;
	buffer[saLen + 4] = 0x80;

	//U0 Gen
	//8 -> passwordLen
	((uint64_t*)(buffer))[15] = (128 + 8) << 3;
	KMU_PBKDF_SALTv1(IPAD_out, (uint64_t*)buffer, dk1);
	KMU_PBKDF_SHA_BLOCK_COREv2(OPAD_out, dk1, 192, dk2);

	for (int j = 0; j < 8; j++)
		out[j] = dk2[j];
	for (int i = 1; i < iteration; i++) {
		KMU_PBKDF_SHA_BLOCK_COREv2(IPAD_out, dk2, 192, dk1);
		KMU_PBKDF_SHA_BLOCK_COREv2(OPAD_out, dk1, 192, dk2);
		
		for (int j = 0; j < 8; j++) {
			out[j] ^= dk2[j];
		}
	}
	if (out[0] == DK[0]) {
		cracking[0] = 1;
	}
}

//6byte cracking Sample
//__global__ void KMU_PBKDF2v2(uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* Target)
//{
//	//Password Word Setting
//	uint64_t pt = 0;
//	uint64_t pt1 = 0;
//	uint32_t flag = blockDim.x * blockIdx.x + threadIdx.x;
//	uint8_t cracking[1];
//	cracking[0] = 1;
//	pt = (((flag) % 128)) | (((flag / 128) % 128) << 8) | (((flag / 16384) % 128) << 16);
//	pt1 = pt;
//	flag = (flag / 2097152) % 128; //(0 <= flag < 16)
//	for (uint64_t i = 0; i < 16384; i++) {
//		pt = pt + ((i % 128) << 24);
//		pt = pt + ((i / 128) << 32);
//		KMU_PKBDF2_Core_v2(pt, 1, iteration, salt, saLen, Target, cracking);
//		pt = pt1;
//	}
//	
//}

//2byte cracking Sample
__global__ void KMU_PBKDF2v3(uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* Target)
{
	//Password Word Setting
	uint64_t pt = 0;
	uint64_t pt1 = 0;
	//uint64_t tt = Target[0];
	uint8_t cracking[1];
	cracking[0] = 0;
	pt = blockIdx.x;
	pt = pt << 56;
	pt1 = threadIdx.x;
	pt = pt + (pt1 << 48);

	KMU_PKBDF2_Core_v2(pt, 1, iteration, salt, saLen, Target, cracking);
	if (cracking[0] == 1) {
		printf("%016llx ", pt);
	}

}

__global__ void KMU_PBKDF2v4(uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* Target)
{
	//Password Word Setting
	uint64_t pt = 0;
	uint64_t pt1 = 0;
	//uint64_t tt = Target[0];
	uint8_t cracking[1];
	cracking[0] = 0;
	uint64_t flag = (blockDim.x * blockIdx.x) + threadIdx.x;

	pt = (flag % 128) << 56;
	pt = pt | (((flag >> 7) & 0x7F) << 48);
	pt = pt | (((flag / 16384) & 0x7F) << 40);
	for (int j = 0; j < 2; j++) {
		pt = pt | ((j & 0x7F) << 32);
		KMU_PKBDF2_Core_v2(pt, 1, iteration, salt, saLen, Target, cracking);
	}
	if (cracking[0] == 1) {
		printf("%016llx ", pt);
	}

}


__global__ void KMU_PBKDF2v5(uint64_t iteration, uint8_t* salt, uint64_t saLen, uint64_t* Target, int stream)
{
	//Password Word Setting
	uint64_t pt = 0;
	uint64_t pt1 = 0;
	//uint64_t tt = Target[0];
	uint8_t cracking[1];
	cracking[0] = 0;
	uint64_t flag = (blockDim.x * blockIdx.x) + threadIdx.x;
	pt = (flag % 128) << 56;
	pt = pt | (((flag >> 7) & 0x7F) << 48);
	pt = pt | (((flag / 16384) & 0x7F) << 40);
	pt = pt | ((stream & 0x7F) << 32);
	KMU_PKBDF2_Core_v2(pt, 1, iteration, salt, saLen, Target, cracking);
	if (cracking[0] == 1) {
		printf("%016llx ", pt);
	}
}

__global__ void Core()
{

}

int main()
{
	uint8_t salt[4];
	uint64_t Target[1];
	hipStream_t stream[2];

	hipEvent_t start, stop;
	hipError_t err;
	float elapsed_time_ms = 0.0f;
	
	salt[0] = 0x73;
	salt[1] = 0x61;
	salt[2] = 0x6c;
	salt[3] = 0x74;
	Target[0] = 0x9D536145420D4242;
	TT[0] = Target[0];
	uint8_t* cuda_salt = NULL;
	uint64_t* cuda_Target = NULL;

	hipMalloc((void**)&cuda_salt, 4);
	hipMalloc((void**)&cuda_Target, 8);

	hipMemcpy(cuda_salt, salt, 4, hipMemcpyHostToDevice);
	hipMemcpy(cuda_Target, Target, 8, hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (int i = 0; i < 1; i++) {
		//KMU_PBKDF2v4 << <4096, 512 >> > (129937, cuda_salt, 4, cuda_Target);
	}hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("Performance : %4.2f/s\n", elapsed_time_ms/1000);

	uint8_t* Salt = NULL;
	uint64_t* TTarget = NULL;
	uint8_t* cuda_salt_s0 = NULL;
	uint8_t* cuda_salt_s1 = NULL;
	uint64_t* cuda_Target0 = NULL;
	uint64_t* cuda_Target1 = NULL;

	hipMalloc((void**)&cuda_salt_s0, 4);
	hipMalloc((void**)&cuda_salt_s1, 4);
	hipMalloc((void**)&cuda_Target0, 8);
	hipMalloc((void**)&cuda_Target1, 8);
	hipHostMalloc((void**)&Salt, 4);
	hipHostMalloc((void**)&TTarget, 8);

	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);

	Salt[0] = 0x73;
	Salt[1] = 0x61;
	Salt[2] = 0x6c;
	Salt[3] = 0x74;
	TTarget[0] = 0x9D536145420D4242;


	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMemcpyAsync(cuda_salt_s0, Salt, 4, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(cuda_Target0, TTarget, 8, hipMemcpyHostToDevice, stream[0]);
	KMU_PBKDF2v5 << <4096, 512, 0, stream[0] >> > (129937, cuda_salt, 4, cuda_Target, 0);
	hipMemcpyAsync(cuda_salt_s1, Salt, 4, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(cuda_Target1, TTarget, 8, hipMemcpyHostToDevice, stream[1]);
	KMU_PBKDF2v5 << <4096, 512, 0, stream[1] >> > (129937, cuda_salt, 4, cuda_Target, 1);
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("Performance : %4.2f/s\n", elapsed_time_ms / 1000);

}